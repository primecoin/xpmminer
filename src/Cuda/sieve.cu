#include "hip/hip_runtime.h"
#define S1RUNS (sizeof(nps_all)/sizeof(uint32_t))
#define NLIFO 4

// for 1024 threads in group
#if (LSIZELOG2 == 10)
__constant__ uint32_t nps_all[] = { 4, 4, 5, 6, 7, 7, 7, 9 }; // 1024 threads per block (default)
#elif (LSIZELOG2 == 9)
__constant__ uint32_t nps_all[] = { 3, 3, 4, 5, 6, 6, 6, 7, 7 }; // 512 threads
#elif (LSIZELOG2 == 8)
__constant__ uint32_t nps_all[] = { 2, 2, 3, 4, 5, 5, 5, 6, 6 }; // 256 threads
#else
#error "Unsupported LSIZELOG2 constant"
#endif

__global__ void sieve(uint32_t *gsieve_all,
                      uint32_t* offset_all,
                      uint2 *primes)
{
  __shared__ uint32_t sieve[SIZE];
  
  const uint32_t id = threadIdx.x;
  const uint32_t stripe = blockIdx.x;
  const uint32_t line = blockIdx.y; 
  const uint32_t entry = SIZE*32*(stripe+STRIPES/2);
  const float fentry = entry;
  
  const uint32_t* offset = &offset_all[PCOUNT*line];
  
  for (uint32_t i = id; i < SIZE; i += LSIZE)
    sieve[i] = 0;
  __syncthreads();
  
  uint32_t poff = 0;

#pragma unroll
  for(int b = 0; b < S1RUNS; b++) {
    uint32_t nps = nps_all[b];
    const uint32_t var = LSIZE >> nps;
    const uint32_t lpoff = id & (var-1);
    uint32_t ip = id >> (LSIZELOG2-nps);

    const uint2 tmp1 = primes[poff+ip];
    const uint32_t prime = tmp1.x;
    const float fiprime = __int_as_float(tmp1.y);

    const uint32_t loffset = offset[poff+ip];
    const uint32_t orb = (loffset >> 31) ^ 0x1;
    uint32_t pos = loffset & 0x7FFFFFFF;

    poff += 1u << nps;
    pos += __umul24((uint32_t)(fentry * fiprime), prime);
      pos -= entry;
    pos += ((int)pos < 0 ? prime : 0);
#if STRIPES > 256
    pos += ((int)pos < 0 ? prime : 0);
#endif
    pos += __umul24(lpoff, prime);

    uint4 vpos = {pos,
                  pos + __umul24(var, prime),
                  pos + __umul24(var*2, prime),
                  pos + __umul24(var*3, prime)};

    if (var*4 >= 32) {
      uint32_t *s1 = &sieve[vpos.x >> 5];
      uint32_t *s2 = &sieve[vpos.y >> 5];
      uint32_t *s3 = &sieve[vpos.z >> 5];
      uint32_t *s4 = &sieve[vpos.w >> 5];
      uint32_t *se = &sieve[SIZE];
      uint32_t bit1 = orb << (vpos.x % 32);
      uint32_t bit2 = orb << (vpos.y % 32);
      uint32_t bit3 = orb << (vpos.z % 32);
      uint32_t bit4 = orb << (vpos.w % 32);
      const uint32_t add = var*4*prime >> 5;
      while (s4 < se) {
        atomicOr(s1, bit1);
        atomicOr(s2, bit2);
        atomicOr(s3, bit3);
        atomicOr(s4, bit4);
        s1 += add;
        s2 += add;
        s3 += add;
        s4 += add;
      }

      if (s1 < se)
        atomicOr(s1, bit1);
      if (s2 < se)
        atomicOr(s2, bit2);
      if (s3 < se)
        atomicOr(s3, bit3);
    } else {


    const uint32_t add = var*4*prime;
    while (vpos.w < SIZE*32) {
      atomicOr(&sieve[vpos.x >> 5], orb << (vpos.x%32));
      atomicOr(&sieve[vpos.y >> 5], orb << (vpos.y%32));
      atomicOr(&sieve[vpos.z >> 5], orb << (vpos.z%32));
      atomicOr(&sieve[vpos.w >> 5], orb << (vpos.w%32));
      vpos.x += add;
      vpos.y += add;
      vpos.z += add;
      vpos.w += add;
    }

    if (vpos.x < SIZE*32)
      atomicOr(&sieve[vpos.x >> 5], orb << (vpos.x%32));
    if (vpos.y < SIZE*32)
      atomicOr(&sieve[vpos.y >> 5], orb << (vpos.y%32));
    if (vpos.z < SIZE*32)
      atomicOr(&sieve[vpos.z >> 5], orb << (vpos.z%32));
    }
  }
  
  const uint2 *pprimes = &primes[id];
  const uint32_t *poffset = &offset[id];
  
  uint32_t plifo[NLIFO];
  uint32_t fiplifo[NLIFO];
  uint32_t olifo[NLIFO];

  for(int i = 0; i < NLIFO; ++i){
    pprimes += LSIZE;
    poffset += LSIZE;
    
    const uint2 tmp = *pprimes;
    plifo[i] = tmp.x;
    fiplifo[i] = tmp.y;
    olifo[i] = *poffset;
  }
  
  uint32_t lpos = 0;
  
#pragma unroll
  for (uint32_t ip = 1; ip < SIEVERANGE3; ++ip) {
    const uint32_t prime = plifo[lpos];
    const float fiprime = __int_as_float(fiplifo[lpos]);
    uint32_t pos = olifo[lpos];
    
    pos += __umul24((uint32_t)(fentry * fiprime), prime);
      pos -= entry;
    pos += ((int)pos < 0 ? prime : 0);
    
    uint32_t index = pos >> 5;
    
    if(ip < SIEVERANGE1){
      uint2 vpos = {pos,
                    pos + prime};
        
      const uint32_t add = 2*prime;
      while (vpos.y < SIZE*32) {
        atomicOr(&sieve[vpos.x >> 5], 1u << (vpos.x%32));
        atomicOr(&sieve[vpos.y >> 5], 1u << (vpos.y%32));
        vpos.x += add;
        vpos.y += add;
      }
        
      if (vpos.x < SIZE*32)
        atomicOr(&sieve[vpos.x >> 5], 1u << (vpos.x % 32));
    } else if (ip < SIEVERANGE2) {
      if(index < SIZE){
        atomicOr(&sieve[index], 1u << (pos%32));
        pos += prime;
        index = pos >> 5;
        if(index < SIZE){
          atomicOr(&sieve[index], 1u << (pos%32));
          pos += prime;
          index = pos >> 5;
          if(index < SIZE){
            atomicOr(&sieve[index], 1u << (pos%32));
          }
        }
      }
    } else if(ip < SIEVERANGE3) {
      if(index < SIZE){
        atomicOr(&sieve[index], 1u << (pos%32));
        pos += prime;
        index = pos >> 5;
        if(index < SIZE){
          atomicOr(&sieve[index], 1u << (pos%32));
        }
      }
    } else {
      if(index < SIZE){
        atomicOr(&sieve[index], 1u << (pos%32));
      }
    }
    
    if(ip+NLIFO < SCOUNT/LSIZE){
      pprimes += LSIZE;
      poffset += LSIZE;
      
      const uint2 tmp = *pprimes;
      plifo[lpos] = tmp.x;
      fiplifo[lpos] = tmp.y;
      olifo[lpos] = *poffset;
    }
    
    lpos++;
    lpos = lpos % NLIFO;
  }

#pragma unroll
  for (uint32_t ip = SIEVERANGE3; ip < SCOUNT/LSIZE; ++ip) {
    const uint32_t prime = plifo[lpos];
    const float fiprime = __int_as_float(fiplifo[lpos]);
    uint32_t pos = olifo[lpos];

    pos += __umul24((uint32_t)(fentry * fiprime), prime);
      pos -= entry;
    pos += ((int)pos < 0 ? prime : 0);

    uint32_t index = pos >> 5;
    if(index < SIZE)
      atomicOr(&sieve[index], 1u << (pos%32));

    if(ip+NLIFO < SCOUNT/LSIZE){
      pprimes += LSIZE;
      poffset += LSIZE;

      const uint2 tmp = *pprimes;
      plifo[lpos] = tmp.x;
      fiplifo[lpos] = tmp.y;
      olifo[lpos] = *poffset;
    }

    lpos++;
    lpos = lpos % NLIFO;
  }

  __syncthreads();
  uint32_t *gsieve = &gsieve_all[SIZE*(STRIPES/2*line + stripe)];
  for (uint32_t i = id; i < SIZE; i += LSIZE)
    gsieve[i] = sieve[i];
}

__global__ void s_sieve(const uint32_t *gsieve1,
                        const uint32_t* gsieve2,
                        fermat_t *found320,
                        fermat_t *found352,
                        uint32_t *fcount,
                        uint32_t hashid,
                        uint32_t hashSize,
                        uint32_t depth)
{
  const uint32_t id = blockIdx.x * blockDim.x + threadIdx.x;

  uint32_t tmp1[WIDTH];
#pragma unroll
  for (int i = 0; i < WIDTH; ++i)
    tmp1[i] = gsieve1[SIZE*STRIPES/2*i + id];

#pragma unroll
  for (int start = 0; start <= WIDTH-TARGET; ++start){
    uint32_t mask = 0;

#pragma unroll
    for (int line = 0; line < TARGET; ++line)
      mask |= tmp1[start+line];

    if (mask != 0xFFFFFFFF) {
      unsigned bit = 31-__clz(~mask);
      unsigned multiplier = bit + id*32 + SIZE*32*STRIPES/2;  // mad24(id, 32u, (unsigned)bit) + SIZE*32*STRIPES/2;
      unsigned maxSize = hashSize + (32-__clz(multiplier)) + start + depth;
      const uint32_t addr = atomicAdd(&fcount[(maxSize <= 320) ? 0 : 1], 1);
      fermat_t *found = (maxSize <= 320) ? found320 : found352;

      fermat_t info;
      info.index = multiplier;
      info.origin = start;
      info.chainpos = 0;
      info.type = 0;
      info.hashid = hashid;
      found[addr] = info;
    }
  }

  uint32_t tmp2[WIDTH];
#pragma unroll
  for (int i = 0; i < WIDTH; ++i)
    tmp2[i] = gsieve2[SIZE*STRIPES/2*i + id];

#pragma unroll
  for (int start = 0; start <= WIDTH-TARGET; ++start){
    uint32_t mask = 0;
#pragma unroll
    for (int line = 0; line < TARGET; ++line)
      mask |= tmp2[start+line];

    if (mask != 0xFFFFFFFF) {
      unsigned bit = 31-__clz(~mask);
      unsigned multiplier = bit + id*32 + SIZE*32*STRIPES/2;  // mad24(id, 32u, (unsigned)bit) + SIZE*32*STRIPES/2;
      unsigned maxSize = hashSize + (32-__clz(multiplier)) + start + depth;
      const uint32_t addr = atomicAdd(&fcount[(maxSize <= 320) ? 0 : 1], 1);
      fermat_t *found = (maxSize <= 320) ? found320 : found352;

      fermat_t info;
      info.index = multiplier;
      info.origin = start;
      info.chainpos = 0;
      info.type = 1;
      info.hashid = hashid;
      found[addr] = info;
    }
  }

  const unsigned bitwinLayers = (TARGET / 2) + (TARGET % 2);
#pragma unroll
  for (int i = 0; i < WIDTH; ++i)
    tmp2[i] |= tmp1[i];
#pragma unroll
  for (int start = 0; start <= WIDTH-bitwinLayers; ++start) {
    uint32_t mask = 0;
#pragma unroll
    for(int line = 0; line < TARGET/2; ++line)
      mask |= tmp2[start+line];

    if(TARGET & 1u)
      mask |= tmp1[start+TARGET/2];

    if (mask != 0xFFFFFFFF) {
      unsigned bit = 31-__clz(~mask);
      unsigned multiplier = bit + id*32 + SIZE*32*STRIPES/2;  // mad24(id, 32u, (unsigned)bit) + SIZE*32*STRIPES/2;
      unsigned maxSize = hashSize + (32-__clz(multiplier)) + start + (depth/2) + (depth&1);
      const uint32_t addr = atomicAdd(&fcount[(maxSize <= 320) ? 0 : 1], 1);
      fermat_t *found = (maxSize <= 320) ? found320 : found352;

      fermat_t info;
      info.index = multiplier;
      info.origin = start;
      info.chainpos = 0;
      info.type = 2;
      info.hashid = hashid;
      found[addr] = info;
    }
  }
}
