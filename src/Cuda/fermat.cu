#include "hip/hip_runtime.h"
#define N 12
#define SCOUNT PCOUNT

__constant__ uint32_t pow2[9] = {1, 2, 4, 8, 16, 32, 64, 128, 256};

__constant__ uint32_t binvert_limb_table[128] = {
  0x01, 0xAB, 0xCD, 0xB7, 0x39, 0xA3, 0xC5, 0xEF,
  0xF1, 0x1B, 0x3D, 0xA7, 0x29, 0x13, 0x35, 0xDF,
  0xE1, 0x8B, 0xAD, 0x97, 0x19, 0x83, 0xA5, 0xCF,
  0xD1, 0xFB, 0x1D, 0x87, 0x09, 0xF3, 0x15, 0xBF,
  0xC1, 0x6B, 0x8D, 0x77, 0xF9, 0x63, 0x85, 0xAF,
  0xB1, 0xDB, 0xFD, 0x67, 0xE9, 0xD3, 0xF5, 0x9F,
  0xA1, 0x4B, 0x6D, 0x57, 0xD9, 0x43, 0x65, 0x8F,
  0x91, 0xBB, 0xDD, 0x47, 0xC9, 0xB3, 0xD5, 0x7F,
  0x81, 0x2B, 0x4D, 0x37, 0xB9, 0x23, 0x45, 0x6F,
  0x71, 0x9B, 0xBD, 0x27, 0xA9, 0x93, 0xB5, 0x5F,
  0x61, 0x0B, 0x2D, 0x17, 0x99, 0x03, 0x25, 0x4F,
  0x51, 0x7B, 0x9D, 0x07, 0x89, 0x73, 0x95, 0x3F,
  0x41, 0xEB, 0x0D, 0xF7, 0x79, 0xE3, 0x05, 0x2F,
  0x31, 0x5B, 0x7D, 0xE7, 0x69, 0x53, 0x75, 0x1F,
  0x21, 0xCB, 0xED, 0xD7, 0x59, 0xC3, 0xE5, 0x0F,
  0x11, 0x3B, 0x5D, 0xC7, 0x49, 0x33, 0x55, 0xFF
};


typedef struct {
  uint32_t index;
  uint32_t hashid;
  uint8_t origin;
  uint8_t chainpos;
  uint8_t type;
  uint8_t reserved;
} fermat_t;

typedef struct {
  uint32_t N_;
  uint32_t SIZE_;
  uint32_t STRIPES_;
  uint32_t WIDTH_;
  uint32_t PCOUNT_;
  uint32_t TARGET_;
  uint32_t LIMIT13_;
  uint32_t LIMIT14_;
  uint32_t LIMIT15_;
} config_t;

__global__ void getconfig(config_t *conf)
{
  config_t c;
  c.N_ = N;
  c.SIZE_ = SIZE;
  c.STRIPES_ = STRIPES;
  c.WIDTH_ = WIDTH;
  c.PCOUNT_ = PCOUNT;
  c.TARGET_ = TARGET;
  c.LIMIT13_ = LIMIT13;
  c.LIMIT14_ = LIMIT14;
  c.LIMIT15_ = LIMIT15;
  *conf = c;
}

__device__ void shr32(uint32_t *data, unsigned size)
{
#pragma unroll
  for (int j = 1; j < size; j++)
    data[j-1] = data[j];
  data[size-1] = 0;
}

__device__ void shl(uint32_t *data, unsigned size, unsigned bits)
{
  #pragma unroll
  for(int i = size-1; i > 0; i--)
    data[i] = (data[i] << bits) | (data[i-1] >> (32-bits));
  
  data[0] = data[0] << bits;
}

__device__ void shr(uint32_t *data, unsigned size, unsigned bits)
{
  #pragma unroll
  for(int i = 0; i < size-1; i++)
    data[i] = (data[i] >> bits) | (data[i+1] << (32-bits));
  data[size-1] = data[size-1] >> bits;
}

__device__ uint32_t invert_limb(uint32_t limb)
{
  uint32_t inv = binvert_limb_table[(limb/2) & 0x7F];
  inv = 2*inv - inv*inv*limb;
  inv = 2*inv - inv*inv*limb;
  return -inv;
}

__device__ uint32_t getFromBitfield(const uint32_t *ptr, unsigned bitOffset, unsigned bitSize)
{
  union {
    uint2 v32;
    uint64_t v64;
  } data;  
  
  unsigned lbitOffset = bitOffset & 0x1F;
  unsigned lLoLimb = bitOffset >> 5;
  unsigned lHiLimb = (bitOffset+bitSize) >> 5;
  data.v32.x = ptr[lLoLimb];
  data.v32.y = (lLoLimb == lHiLimb) ? 0 : ptr[lHiLimb];
  data.v32.x = lLoLimb == 0 ? data.v32.x - 1 : data.v32.x;
  return (data.v64 >> lbitOffset) & ((1 << bitSize) - 1);
}

__device__ void redcify352(unsigned shiftCount,
                           const uint32_t *quotient,
                           const uint32_t *limbs,
                           uint32_t *result,
                           uint32_t windowSize)
{
  uint32_t q[8];
  q[0] = quotient[0];
  q[1] = quotient[1];
  q[2] = quotient[2];
  q[3] = quotient[3];
  q[4] = quotient[4];
  q[5] = quotient[5];
  q[6] = quotient[6];
  q[7] = quotient[7];  

  const unsigned pow2ws = pow2[windowSize];  
  
  for (unsigned  i = 0, ie = (pow2ws-shiftCount)/32; i < ie; i++)
    shr32(q, 8);
  if ((pow2ws-shiftCount) % 32)
    shr(q, 8, (pow2ws-shiftCount) % 32);

  if (windowSize == 5)
    mulProductScan352to96(result, limbs, q);
  else if (windowSize == 6)
    mulProductScan352to128(result, limbs, q);
  else if (windowSize == 7)
    mulProductScan352to192(result, limbs, q);
  
  // substract 2^(384+shiftCount) - q*R
  for (unsigned i = 0; i < 11; i++)
    result[i] = ~result[i];
  result[0]++;
}

__device__ void FermatTest352(const uint32_t *e, uint32_t *redcl)
{
  const int windowSize = 7;    
  uint32_t inverted = invert_limb(e[0]);  
  uint32_t q[8] = {0, 0, 0, 0, 0, 0, 0, 0};
  int remaining = divide512to352reg(0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1,
                    e[0], e[1], e[2], e[3], e[4], e[5], e[6], e[7], e[8], e[9], e[10],
                    &q[0], &q[1], &q[2], &q[3], &q[4], &q[5], &q[6], &q[7]);
  remaining--;

  
  // Retrieve of "2" in Montgomery representation
  redcify352(1, q, e, redcl, windowSize);  

  while (remaining > 0) {
    int size = min(remaining, windowSize);
    uint32_t index = getFromBitfield(e, remaining-size, size);
    
    uint32_t m[11];
    for (unsigned i = 0; i < size; i++)
      monSqr352(redcl, e, inverted);
    
    redcify352(index, q, e, m, windowSize);    
    monMul352(redcl, m, e, inverted);
    remaining -= windowSize;
  }
  
  redcHalf352(redcl, e, inverted);
}

__device__ void redcify320(unsigned shiftCount,
                           const uint32_t *quotient,
                           const uint32_t *limbs,
                           uint32_t *result,
                           uint32_t windowSize)
{
  uint32_t q[8];
  q[0] = quotient[0];
  q[1] = quotient[1];
  q[2] = quotient[2];
  q[3] = quotient[3];
  q[4] = quotient[4];
  q[5] = quotient[5];
  q[6] = quotient[6];
  q[7] = quotient[7];  
  
  const unsigned pow2ws = pow2[windowSize];   
  for (unsigned  i = 0, ie = (pow2ws-shiftCount)/32; i < ie; i++)
    shr32(q, 8);
  if ((pow2ws-shiftCount) % 32)
    shr(q, 8, (pow2ws-shiftCount) % 32);

  if (windowSize == 5)
    mulProductScan320to96(result, limbs, q);  
  else if (windowSize == 6)
    mulProductScan320to128(result, limbs, q);
  else if (windowSize == 7)
    mulProductScan320to192(result, limbs, q);
  
  // substract 2^(384+shiftCount) - q*R
  for (unsigned i = 0; i < 10; i++)
    result[i] = ~result[i];
  result[0]++;
}

__device__ void FermatTest320(const uint32_t *e, uint32_t *redcl)
{
  const int windowSize = 7;  
  uint32_t inverted = invert_limb(e[0]);  
  uint32_t q[8] = {0, 0, 0, 0, 0, 0, 0, 0};
  int remaining = divide480to320reg(0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1,
                    e[0], e[1], e[2], e[3], e[4], e[5], e[6], e[7], e[8], e[9],
                    &q[0], &q[1], &q[2], &q[3], &q[4], &q[5], &q[6], &q[7]);
  remaining--;
  
  // Retrieve of "2" in Montgomery representation
  redcify320(1, q, e, redcl, windowSize);

  while (remaining > 0) {
    int size = min(remaining, windowSize);
    uint32_t index = getFromBitfield(e, remaining-size, size);
    
    uint32_t m[10];
    for (unsigned i = 0; i < size; i++)
      monSqr320(redcl, e, inverted);
    
    redcify320(index, q, e, m, windowSize);
    monMul320(redcl, m, e, inverted);     
    remaining -= windowSize;
  }
  
  redcHalf320(redcl, e, inverted);
}

__device__ bool fermat352(const uint32_t *p)
{
  uint32_t modpowl[11];
  FermatTest352(p, modpowl);
  
  uint32_t result = modpowl[0] - 1;
  result |= modpowl[1];
  result |= modpowl[2];
  result |= modpowl[3];
  result |= modpowl[4];
  result |= modpowl[5];
  result |= modpowl[6];
  result |= modpowl[7];
  result |= modpowl[8];
  result |= modpowl[9];
  result |= modpowl[10];  
  return result == 0;
}

__device__ bool fermat320(const uint32_t *p)
{
  uint32_t modpowl[10];  
  FermatTest320(p, modpowl);
  
  uint32_t result = modpowl[0] - 1;
  result |= modpowl[1];
  result |= modpowl[2];
  result |= modpowl[3];
  result |= modpowl[4];
  result |= modpowl[5];
  result |= modpowl[6];
  result |= modpowl[7];
  result |= modpowl[8];
  result |= modpowl[9];
  return result == 0;  
}

__device__ uint32_t int_invert(uint32_t a, uint32_t nPrime)
{
    // Extended Euclidean algorithm to calculate the inverse of a in finite field defined by nPrime
    int rem0 = nPrime, rem1 = a % nPrime, rem2;
    int aux0 = 0, aux1 = 1, aux2;
    int quotient, inverse;
    
    while (1)
    {
        if (rem1 <= 1)
        {
            inverse = aux1;
            break;
        }
        
        rem2 = rem0 % rem1;
        quotient = rem0 / rem1;
        aux2 = -quotient * aux1 + aux0;
        
        if (rem2 <= 1)
        {
            inverse = aux2;
            break;
        }
        
        rem0 = rem1 % rem2;
        quotient = rem1 / rem2;
        aux0 = -quotient * aux2 + aux1;
        
        if (rem0 <= 1)
        {
            inverse = aux0;
            break;
        }
        
        rem1 = rem2 % rem0;
        quotient = rem2 / rem0;
        aux1 = -quotient * aux0 + aux2;
    }
    
    return (inverse + nPrime) % nPrime;
}

__global__ void setup_fermat(uint32_t *fprimes,
                            const fermat_t *info_all,
                            uint32_t *hash)
{
  const uint32_t id = blockIdx.x * blockDim.x + threadIdx.x;
  const uint32_t gsize = gridDim.x * blockDim.x;
  const fermat_t info = info_all[id];
  
  uint32_t h[11];
  uint32_t m[11];

  uint32_t *H = &hash[info.hashid*N]; 
#pragma unroll
  for (unsigned i = 0; i < 11; i++)
    h[i] = H[i];

  uint32_t line = info.origin;
  if(info.type < 2)
    line += info.chainpos;
  else
    line += info.chainpos/2;

  uint32_t modifier = (info.type == 1 || (info.type == 2 && (info.chainpos & 1))) ? 1 : -1;

  mulProductScan352to32(m, h, info.index);
  if (line)
    shl(m, 11, line);
  m[0] += modifier;
  
#pragma unroll
  for (unsigned i = 0; i < 11; i++)
    fprimes[gsize*i + id] = m[i];
}


__global__ void fermat_kernel(uint8_t *result, const uint32_t *fprimes)
{
  const uint32_t id = blockIdx.x * blockDim.x + threadIdx.x;
  const uint32_t gsize = gridDim.x * blockDim.x;  
  uint32_t e[11];
  
#pragma unroll
  for (unsigned i = 0; i < 11; i++)
    e[i] = fprimes[gsize*i + id];
  
  result[id] = fermat352(e);
}

__global__ void fermat_kernel320(uint8_t *result, const uint32_t *fprimes)
{
  const uint32_t id = blockIdx.x * blockDim.x + threadIdx.x;
  const uint32_t gsize = gridDim.x * blockDim.x;  
  uint32_t e[10];
  
#pragma unroll
  for (unsigned i = 0; i < 10; i++)
    e[i] = fprimes[gsize*i + id];  
  
  result[id] = fermat320(e);
}



__global__ void check_fermat(fermat_t *info_out,
                             uint32_t *count,
                             fermat_t *info_fin_out,
                             uint32_t *count_fin,
                             const uint8_t *results,
                             const fermat_t *info_in,
                             uint32_t depth)
{
	
	const uint32_t id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(results[id] == 1){
		
		fermat_t info = info_in[id];
		info.chainpos++;
		
		if(info.chainpos < depth){
			
			const uint32_t i = atomicAdd(count, 1);
			info_out[i] = info;
			
		}else{
			
			const uint32_t i = atomicAdd(count_fin, 1);
			info_fin_out[i] = info;
			
		}
		
	}
	
}


__device__ uint32_t mod32(uint32_t *data, unsigned size, uint32_t *modulos, uint32_t divisor)
{
  uint64_t acc = data[0];
  for (unsigned i = 1; i < size; i++)
    acc += (uint64_t)modulos[i-1] * (uint64_t)data[i];
  return acc % divisor;
}

__global__ void setup_sieve(uint32_t *offset1,
                            uint32_t *offset2,
                            const uint32_t *vPrimes,
                            uint32_t *hash,
                            uint32_t hashid,
                            uint32_t *modulos)
{
  
  const uint32_t id = blockIdx.x * blockDim.x + threadIdx.x;
  const uint32_t nPrime = vPrimes[id];
  
  uint32_t tmp[N];
#pragma unroll
  for(int i = 0; i < N; ++i)
    tmp[i] = hash[hashid*N + i];
  
  uint32_t localModulos[N-2];
#pragma unroll
  for (unsigned i = 0; i < N-2; i++)
    localModulos[i] = modulos[PCOUNT*i + id];
  const uint32_t nFixedFactorMod = mod32(tmp, N-1, localModulos, nPrime);
  
  if(nFixedFactorMod == 0){
    for(uint32_t line = 0; line < WIDTH; ++line){
      offset1[PCOUNT*line + id] = 0; //1u << 31;
      offset2[PCOUNT*line + id] = 0; //1u << 31;
    }
    return;
    
  }
  
  uint32_t nFixedInverse = int_invert(nFixedFactorMod, nPrime);
  for(uint32_t layer = 0; layer < WIDTH; ++layer) {
    offset1[PCOUNT*layer + id] = nFixedInverse;
    offset2[PCOUNT*layer + id] = nPrime - nFixedInverse;
    nFixedInverse = (nFixedInverse & 0x1) ?
    (nFixedInverse + nPrime) / 2 : nFixedInverse / 2;
  }    
}
