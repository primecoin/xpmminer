#include "hip/hip_runtime.h"
__global__ void squareBenchmark320(uint32_t *m1,
                                   uint32_t *out,
                                   unsigned elementsNum)
{
#define OperandSize 10
#define GmpOperandSize 10
  unsigned globalSize = gridDim.x * blockDim.x;
  unsigned globalId = blockIdx.x * blockDim.x + threadIdx.x;
  for (unsigned i = globalId; i < elementsNum; i += globalSize) {
    uint32_t op1[OperandSize];
    for (unsigned j = 0; j < OperandSize; j++)
      op1[j] = m1[i*GmpOperandSize + j];  

      uint32_t result[20];

    for (unsigned repeatNum = 0; repeatNum < 512; repeatNum++) {
      sqrProductScan320(result, op1);
      for (unsigned k = 0; k < 10; k++)
        op1[k] = result[k+10];
    }

    for (unsigned j = 0; j < OperandSize*2; j++)
      out[i*OperandSize*2 + j] = result[j];
  }
#undef GmpOperandSize
#undef OperandSize
}

__global__ void squareBenchmark352(uint32_t *m1,
                                   uint32_t *out,
                                   unsigned elementsNum)
{
#define OperandSize 11
#define GmpOperandSize 12  
  unsigned globalSize = gridDim.x * blockDim.x;
  unsigned globalId = blockIdx.x * blockDim.x + threadIdx.x;
  for (unsigned i = globalId; i < elementsNum; i += globalSize) {
    uint32_t op1[OperandSize];
    for (unsigned j = 0; j < OperandSize; j++)
      op1[j] = m1[i*GmpOperandSize + j];  

    uint32_t result[22];
    for (unsigned repeatNum = 0; repeatNum < 512; repeatNum++) {
      sqrProductScan352(result, op1);
      for (unsigned k = 0; k < 11; k++)
        op1[k] = result[k+11];      
    }

    for (unsigned j = 0; j < OperandSize*2; j++)
      out[i*OperandSize*2 + j] = result[j];
  }
#undef GmpOperandSize
#undef OperandSize
}


__global__ void multiplyBenchmark320(uint32_t *m1,
                                     uint32_t *m2,
                                     uint32_t *out,
                                     unsigned elementsNum)
{
#define OperandSize 10
#define GmpOperandSize 10  
  unsigned globalSize = gridDim.x * blockDim.x;
  unsigned globalId = blockIdx.x * blockDim.x + threadIdx.x;
  for (unsigned i = globalId; i < elementsNum; i += globalSize) {
    uint32_t op1[OperandSize];
    uint32_t op2[OperandSize];
    for (unsigned j = 0; j < OperandSize; j++)
      op1[j] = m1[i*GmpOperandSize + j];
    for (unsigned j = 0; j < OperandSize; j++)
      op2[j] = m2[i*GmpOperandSize + j];    
    
    uint32_t result[20];

    for (unsigned repeatNum = 0; repeatNum < 512; repeatNum++) {
      mulProductScan320to320(result, op1, op2);
      for (unsigned k = 0; k < 10; k++)
        op1[k] = result[k+10];      
    }

    for (unsigned j = 0; j < OperandSize*2; j++)
      out[i*OperandSize*2 + j] = result[j];
  }
#undef GmpOperandSize
#undef OperandSize
}

__global__ void multiplyBenchmark352(uint32_t *m1,
                                     uint32_t *m2,
                                     uint32_t *out,
                                     unsigned elementsNum)
{
#define OperandSize 11
#define GmpOperandSize 12  
  unsigned globalSize = gridDim.x * blockDim.x;
  unsigned globalId = blockIdx.x * blockDim.x + threadIdx.x;
  for (unsigned i = globalId; i < elementsNum; i += globalSize) {
    uint32_t op1[OperandSize];
    uint32_t op2[OperandSize];
    for (unsigned j = 0; j < OperandSize; j++)
      op1[j] = m1[i*GmpOperandSize + j];
    for (unsigned j = 0; j < OperandSize; j++)
      op2[j] = m2[i*GmpOperandSize + j];    
    
    uint32_t result[22];    

    for (unsigned repeatNum = 0; repeatNum < 512; repeatNum++) {
      mulProductScan352to352(result, op1, op2);
      for (unsigned k = 0; k < 11; k++)
        op1[k] = result[k+11];         
    }

    for (unsigned j = 0; j < OperandSize*2; j++)
      out[i*OperandSize*2 + j] = result[j];
  }
  
#undef GmpOperandSize
#undef OperandSize
}

__global__ void fermatTestBenchMark320(uint32_t *numbers,
                                       uint32_t *out,
                                       unsigned elementsNum)
{
#define OperandSize 10  
  unsigned globalSize = gridDim.x * blockDim.x;
  unsigned globalId = blockIdx.x * blockDim.x + threadIdx.x;
  for (unsigned i = globalId; i < elementsNum; i += globalSize) {
    uint32_t result[10];
    uint32_t lNumbers[OperandSize];
    for (unsigned j = 0; j < OperandSize; j++)
      lNumbers[j] = numbers[i*OperandSize+j];

    FermatTest320(lNumbers, result);
   
    for (unsigned j = 0; j < OperandSize; j++)
      out[i*OperandSize + j] = result[j];  
  }
#undef OperandSize
}


__global__ void fermatTestBenchMark352(uint32_t *numbers,
                                       uint32_t *out,
                                       unsigned elementsNum)
{
#define OperandSize 11  
  unsigned globalSize = gridDim.x * blockDim.x;
  unsigned globalId = blockIdx.x * blockDim.x + threadIdx.x;
  for (unsigned i = globalId; i < elementsNum; i += globalSize) {
    uint32_t result[11];
    uint32_t lNumbers[OperandSize];
    for (unsigned j = 0; j < OperandSize; j++)
      lNumbers[j] = numbers[i*OperandSize+j];

    FermatTest352(lNumbers, result);
 
    for (unsigned j = 0; j < OperandSize; j++)
      out[i*OperandSize + j] = result[j];  
  }
#undef OperandSize
}
