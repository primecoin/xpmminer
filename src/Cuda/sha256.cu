#include "hip/hip_runtime.h"


__constant__ uint32_t k[] = {
   0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
   0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
   0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
   0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
   0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
   0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
   0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
   0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2 };

__constant__ uint32_t h_init[] = { 0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a, 0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19 };

#define HashPrimorial 16

#define Zrotr(a, b) ((a << b) | (a >> (32 - b)))
#define Ch(x, y, z) (z ^ (x & (y ^ z)))
#define Ma(x, y, z) ((x & z) | (y & (x | z)))

#define ZR25(n) ((Zrotr((n), 25) ^ Zrotr((n), 14) ^ ((n) >> 3U)))
#define ZR15(n) ((Zrotr((n), 15) ^ Zrotr((n), 13) ^ ((n) >> 10U)))
#define ZR26(n) ((Zrotr((n), 26) ^ Zrotr((n), 21) ^ Zrotr((n), 7)))
#define ZR30(n) ((Zrotr((n), 30) ^ Zrotr((n), 19) ^ Zrotr((n), 10)))

__constant__ uint32_t indexesOne[] = { 1, 2, 3, 5, 6 };
__constant__ uint32_t divisors24one[] = { 3, 5, 7, 13, 17 };


__constant__ uint32_t indexes[] = { 4, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19 };
__constant__ uint32_t divisors24[] = { 11, 19, 23, 29, 31, 37, 41, 43, 47, 53, 59, 61, 67, 71 };

__constant__ uint32_t modulos24one[] = {
  0x1, 0x1, 0x1, 0x1, 0x1, 0x1, 0x1, 0x1, 0x1, 0x1, 0x1
};

__constant__ uint32_t modulos24[] = {
  0x5, 0x3, 0x4, 0x9, 0x1, 0x5, 0x3, 0x4, 0x9, 0x1, 0x5,            // 11
  0x7, 0xb, 0x1, 0x7, 0xb, 0x1, 0x7, 0xb, 0x1, 0x7, 0xb,            // 19
  0x4, 0x10, 0x12, 0x3, 0xc, 0x2, 0x8, 0x9, 0xd, 0x6, 0x1,          // 23
  0x14, 0x17, 0x19, 0x7, 0x18, 0x10, 0x1, 0x14, 0x17, 0x19, 0x7,    // 29
  0x10, 0x8, 0x4, 0x2, 0x1, 0x10, 0x8, 0x4, 0x2, 0x1, 0x10,         // 31
  0xa, 0x1a, 0x1, 0xa, 0x1a, 0x1, 0xa, 0x1a, 0x1, 0xa, 0x1a,        // 37
  0x10, 0xa, 0x25, 0x12, 0x1, 0x10, 0xa, 0x25, 0x12, 0x1, 0x10,     // 41
  0x23, 0x15, 0x4, 0xb, 0x29, 0x10, 0x1, 0x23, 0x15, 0x4, 0xb,      // 43
  0x2, 0x4, 0x8, 0x10, 0x20, 0x11, 0x22, 0x15, 0x2a, 0x25, 0x1b,    // 47
  0xd, 0xa, 0x18, 0x2f, 0x1c, 0x2e, 0xf, 0x24, 0x2c, 0x2a, 0x10,    // 53 *
  0x23, 0x2d, 0x29, 0x13, 0x10, 0x1d, 0xc, 0x7, 0x9, 0x14, 0x33,    // 59
  0x14, 0x22, 0x9, 0x3a, 0x1, 0x14, 0x22, 0x9, 0x3a, 0x1, 0x14,     // 61
  0xe, 0x3e, 0x40, 0x19, 0xf, 0x9, 0x3b, 0x16, 0x28, 0x18, 0x1,     // 67 *
  0x3a, 0x1b, 0x4, 0x13, 0x25, 0x10, 0x5, 0x6, 0x40, 0x14, 0x18,    // 71
};

__constant__ uint32_t multipliers32one[] = {
   0xaaaaaaab,     // 3
   0x66666667,     // 5
   0x92492493,     // 7
   0x4ec4ec4f,     // 13
   0x78787879,     // 17
};

__constant__ uint32_t multipliers32[] = {
  0x2e8ba2e9,      // 11
  0x6bca1af3,      // 19
  0xb21642c9,      // 23
  0x8d3dcb09,      // 29
  0x84210843,      // 31
  0xdd67c8a7,      // 37
  0x63e7063f,      // 41
  0x2fa0be83,      // 43
  0xae4c415d,      // 47
  0x4d4873ed,      // 53 *
  0x22b63cbf,      // 59
  0x4325c53f,      // 61
  0x7a44c6b,       // 67 *
  0xe6c2b449       // 71
};

__constant__ uint32_t offsets32one[] = {
  1,               // 3
  1,               // 5
  2,               // 7
  2,               // 13
  3                // 17
};

__constant__ uint32_t offsets32[] = {
  1,               // 11
  3,               // 19
  4,               // 23
  4,               // 29
  4,               // 31
  5,               // 37
  4,               // 41
  3,               // 43
  5,               // 47
  4,               // 53 *
  3,               // 59
  4,               // 61
  1,               // 67 *
  6                // 71
};

// * using 24-bit arithmetic with primes 53,67 can produce wrong results!

__constant__ unsigned gPrimes[] = {
  2,3,5,7,11,13,17,19,23,29,31,37,41,43,47,53,59,61,67,71,73
};

__device__ uint32_t sum24(const uint32_t *data, unsigned size, uint32_t *moddata)
{
  unsigned size24 = size*32; size24 += size24 % 24 ? 24 - size24%24 : 0;
  
  uint32_t acc = data[0] & 0x00FFFFFF;
#pragma unroll
  for (unsigned i = 0, bitPos = 24; bitPos < size24; bitPos += 24, i++) {
    uint64_t v64 = *(uint64_t*)(data+bitPos/32) >> (bitPos%32);
    acc += __umul24(v64 &  0xFFFFFF, moddata[i]);
  }
  
  return acc;
}

__device__ unsigned check24(uint32_t X, uint32_t divisor, uint32_t inversedMultiplier, unsigned offset)
{
  return X == divisor*(__umulhi(X, inversedMultiplier) >> offset);
}

__device__ unsigned divisionCheck24(const uint32_t *data,
                                    unsigned size,
                                    uint32_t divisor,
                                    uint32_t *moddata,
                                    uint32_t inversedMultiplier,
                                    unsigned offset)
{
  return check24(sum24(data, size, moddata), divisor, inversedMultiplier, offset);
}


__device__ uint32_t sha2_pack(uint32_t val) {
  
  return ((val & 0xFF) << 24) | ((val & 0xFF00) << 8) | ((val & 0xFF0000) >> 8) | ((val & 0xFF000000) >> 24);
  
}

__device__ void sha256(const uint32_t *msg, uint32_t *s)
{
#define ROUND(num) {\
  const uint32_t temp1 = h + ZR26(e) + Ch(e, f, g) + k[num] + w[num];\
  const uint32_t temp2 = ZR30(a) + Ma(a, b, c);\
  h = g;\
  g = f;\
  f = e;\
  e = d + temp1;\
  d = c;\
  c = b;\
  b = a;\
  a = temp1 + temp2;\
}  
  
  uint32_t w[64];
  
#pragma unroll  
  for(int i = 0; i < 16; ++i)
    w[i] = msg[i];
  
#pragma unroll  
  for(int i = 16; i < 64; ++i){
    
    const uint32_t s0 = ZR25(w[i-15]);
    const uint32_t s1 = ZR15(w[i-2]);
    w[i] = w[i-16] + s0 + w[i-7] + s1;
    
  }
  
  uint32_t a = s[0];
  uint32_t b = s[1];
  uint32_t c = s[2];
  uint32_t d = s[3];
  uint32_t e = s[4];
  uint32_t f = s[5];
  uint32_t g = s[6];
  uint32_t h = s[7];
  


  ROUND(0)
  ROUND(1) 
  ROUND(2)
  ROUND(3)
  ROUND(4)
  ROUND(5)
  ROUND(6)
  ROUND(7)  
  ROUND(8)  
  ROUND(9)  
  ROUND(10)  
  ROUND(11)  
  ROUND(12)  
  ROUND(13)  
  ROUND(14)  
  ROUND(15)    
  
  ROUND(16)
  ROUND(17)
  ROUND(18)
  ROUND(19)
  ROUND(20)
  ROUND(21)
  ROUND(22)
  ROUND(23)  
  ROUND(24)  
  ROUND(25)  
  ROUND(26)  
  ROUND(27)  
  ROUND(28)  
  ROUND(29)  
  ROUND(30)  
  ROUND(31)    

  ROUND(32)
  ROUND(33)
  ROUND(34)
  ROUND(35)
  ROUND(36)
  ROUND(37)
  ROUND(38)
  ROUND(39)  
  ROUND(40)  
  ROUND(41)  
  ROUND(42)  
  ROUND(43)  
  ROUND(44)  
  ROUND(45)  
  ROUND(46)  
  ROUND(47)    

  ROUND(48)
  ROUND(49)
  ROUND(50)
  ROUND(51)
  ROUND(52)
  ROUND(53)
  ROUND(54)
  ROUND(55)  
  ROUND(56)  
  ROUND(57)  
  ROUND(58)  
  ROUND(59)  
  ROUND(60)  
  ROUND(61)  
  ROUND(62)  
  ROUND(63)      
  
  s[0] += a;
  s[1] += b;
  s[2] += c;
  s[3] += d;
  s[4] += e;
  s[5] += f;
  s[6] += g;
  s[7] += h;
  
#undef ROUND
}

__device__ void sha256UsePrecalc(const uint32_t *msg,
                                 uint32_t *s,
                                 const uint32_t *WData, int WSize,
                                 const uint32_t *new1Data, int new1Size,
                                 const uint32_t *new2Data, int new2Size,
                                 const uint32_t *temp2Data, int tmp2Size)
{
#define ROUND(num) {\
  const uint32_t temp1 = h + ZR26(e) + Ch(e, f, g) + k[num] + w[num];\
  const uint32_t temp2 = ZR30(a) + Ma(a, b, c);\
  h = g;\
  g = f;\
  f = e;\
  if (num < new2Size)\
    e = new2Data[num];\
  else\
    e = d + temp1;\
  d = c;\
  c = b;\
  b = a;\
  if (num < new1Size)\
    a = new1Data[num];\
  else if (num < tmp2Size)\
    a = temp1 + temp2Data[num];\
  else\
    a = temp1 + temp2;\
  }
  
  uint32_t w[64];
  
#pragma unroll  
  for(int i = 0; i < 16; ++i)
    w[i] = msg[i];
  
#pragma unroll  
  for(int i = 16; i < 64; ++i){
    
    const uint32_t s0 = ZR25(w[i-15]);
    const uint32_t s1 = ZR15(w[i-2]);
    w[i] = w[i-16] + s0 + w[i-7] + s1;
    
  }
  
  uint32_t a = s[0];
  uint32_t b = s[1];
  uint32_t c = s[2];
  uint32_t d = s[3];
  uint32_t e = s[4];
  uint32_t f = s[5];
  uint32_t g = s[6];
  uint32_t h = s[7];
  


  ROUND(0)
  ROUND(1)
  ROUND(2)
  ROUND(3)
  ROUND(4)
  ROUND(5)
  ROUND(6)
  ROUND(7)  
  ROUND(8)  
  ROUND(9)  
  ROUND(10)  
  ROUND(11)  
  ROUND(12)  
  ROUND(13)  
  ROUND(14)  
  ROUND(15)    
  
  ROUND(16)
  ROUND(17)
  ROUND(18)
  ROUND(19)
  ROUND(20)
  ROUND(21)
  ROUND(22)
  ROUND(23)  
  ROUND(24)  
  ROUND(25)  
  ROUND(26)  
  ROUND(27)  
  ROUND(28)  
  ROUND(29)  
  ROUND(30)  
  ROUND(31)    

  ROUND(32)
  ROUND(33)
  ROUND(34)
  ROUND(35)
  ROUND(36)
  ROUND(37)
  ROUND(38)
  ROUND(39)  
  ROUND(40)  
  ROUND(41)  
  ROUND(42)  
  ROUND(43)  
  ROUND(44)  
  ROUND(45)  
  ROUND(46)  
  ROUND(47)    

  ROUND(48)
  ROUND(49)
  ROUND(50)
  ROUND(51)
  ROUND(52)
  ROUND(53)
  ROUND(54)
  ROUND(55)  
  ROUND(56)  
  ROUND(57)  
  ROUND(58)  
  ROUND(59)  
  ROUND(60)  
  ROUND(61)  
  ROUND(62)  
  ROUND(63)      
  
  s[0] += a;
  s[1] += b;
  s[2] += c;
  s[3] += d;
  s[4] += e;
  s[5] += f;
  s[6] += g;
  s[7] += h;
  
#undef ROUND  
}

#define select(a, b, c) ((c) ? (b) : (a))

__global__ void bhashmodUsePrecalc(uint32_t nonceOffset,
                                   uint32_t *found,
                                   uint32_t *fcount,
                                   uint32_t *resultPrimorial,
                                   uint32_t *midstate,
                                   uint32_t merkle,
                                   uint32_t time,
                                   uint32_t nbits,
                                   uint32_t W0,
                                   uint32_t W1,
                                   uint32_t new1_0,
                                   uint32_t new1_1,
                                   uint32_t new1_2,
                                   uint32_t new2_0,
                                   uint32_t new2_1,
                                   uint32_t new2_2,
                                   uint32_t temp2_3)
{
  const uint32_t id = blockIdx.x * blockDim.x + threadIdx.x + nonceOffset;
  
  uint32_t msg[16];
  msg[0] = merkle;
  msg[1] = time;
  msg[2] = nbits;
  msg[3] = sha2_pack(id);
  msg[4] = sha2_pack(0x80);
  
  #pragma unroll  
  for(int i = 5; i < 15; ++i)
    msg[i] = 0;
  msg[15] = 640;
  
  uint32_t state[9];
#pragma unroll
  for(int i = 0; i < 8; ++i)
    state[i] = midstate[i];
  
  uint32_t W[2] = {W0, W1};
  uint32_t new1[3] = {new1_0, new1_1, new1_2};
  uint32_t new2[3] = {new2_0, new2_1, new2_2};
  uint32_t temp2[4] = {0, 0, 0, temp2_3};  
  
  sha256UsePrecalc(msg, state, W, 2, new1, 3, new2, 3, temp2, 4);
  
  #pragma unroll  
  for(int i = 0; i < 8; ++i)
    msg[i] = state[i];
  msg[8] = sha2_pack(0x80);
  msg[15] = 256;
  
  #pragma unroll  
  for(int i = 0; i < 8; ++i)
    state[i] = h_init[i];
  
  sha256(msg, state);
  for(int i = 0; i < 8; ++i)
    state[i] = sha2_pack(state[i]);
  
  if (state[7] & (1u << 31)) {
    uint32_t count = !(state[0] & 0x1);
    uint32_t primorialBitField = count;
    state[8] = 0;
    
    {
      uint32_t acc = sum24(state, 8, modulos24one);
      #pragma unroll
      for (unsigned i = 0; i < 5; i++) {
        unsigned isDivisor = check24(acc, divisors24one[i], multipliers32one[i], offsets32one[i]);
        primorialBitField |= (isDivisor << indexesOne[i]);
        count += isDivisor;
      }
    }
    
    unsigned lastBit = 0;
    #pragma unroll    
    for (unsigned i = 0; i < HashPrimorial-5; i++) {
      unsigned isDivisor =
      divisionCheck24(state, 8, divisors24[i], &modulos24[i*11], multipliers32[i], offsets32[i]);
      primorialBitField |= (isDivisor << indexes[i]);
      lastBit = isDivisor ? i+5 : lastBit;
    }
    
    uint32_t prod13l = 1;
    for (unsigned i = 0; i < 8; i++)
      prod13l = mul24(prod13l, select(gPrimes[i], 1u, primorialBitField & (1u << i)));
    prod13l *= select(gPrimes[8], 1u, primorialBitField & (1u << 8));
    
    uint64_t prod13 = prod13l;
    for (unsigned i = 9; i < 14; i++)
      prod13 *= select(gPrimes[i], 1u, primorialBitField & (1u << i));
    
    uint64_t prod14 = prod13 * select(gPrimes[14], 1u, primorialBitField & (1u << 14));
    uint64_t prod15 = prod14 * select(gPrimes[15], 1u, primorialBitField & (1u << 15));
    
    int p13isValid = ((64-__clzll(prod13)) < LIMIT13);
    
    int p14Unique = !(p13isValid & (prod14 == prod13));
    int p14isValid = ((64-__clzll(prod14)) < LIMIT14) & p14Unique;
    
    int p15Unique = !(p13isValid & (prod15 == prod13)) & !(p14isValid & (prod15 == prod14));
    int p15isValid = ((64-__clzll(prod15)) < LIMIT15) & p15Unique;
    
    if (p13isValid) {
      const uint32_t index = atomicAdd(fcount, 1);
      resultPrimorial[index] = (primorialBitField & 0xFFFF) | (13u << 16);
      found[index] = id;
    }
    
    if (p14isValid) {
      const uint32_t index = atomicAdd(fcount, 1);
      resultPrimorial[index] = (primorialBitField & 0xFFFF) | (14u << 16);
      found[index] = id;
    }
    
    if (p15isValid) {
      const uint32_t index = atomicAdd(fcount, 1);
      resultPrimorial[index] = (primorialBitField & 0xFFFF) | (15u << 16);
      found[index] = id;
    }    
  }
}
